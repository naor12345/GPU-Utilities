
#include <hip/hip_runtime.h>
#define gerror( err ) (c_HandleError( err, __FILE__, __LINE__ ))

// 打印错误信息与位置
void c_HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d, %d\n", hipGetErrorString(err),
			file, line, err);
		getchar();
		exit(EXIT_FAILURE);
	}
}